#include "hip/hip_runtime.h"
#include "slice/slice.hpp"
#include "common/check.hpp"
#include <cmath>

static __global__ void slice_kernel(
  const uchar3* __restrict__ image,
  uchar3* __restrict__ outs,
  const int width,
  const int height,
  const int slice_width,
  const int slice_height,
  const int slice_num_h,
  const int slice_num_v,
  const int* __restrict__ slice_start_point)
{
    const int slice_idx = blockIdx.z;
    // printf("%d\n", slice_idx);
    const int start_x = slice_start_point[slice_idx * 2];
    const int start_y = slice_start_point[slice_idx * 2 + 1];

    // 当前像素在切片内的相对位置
    const int x = blockIdx.x * blockDim.x + threadIdx.x;
    const int y = blockIdx.y * blockDim.y + threadIdx.y;
    if(x >= slice_width| y >= slice_height) 
    {
        // printf("i, j : %d,%d\t x, y : %d, %d \t w, h : %d, %d\n", i, j, x, y, sdx_end - sdx_start, sdy_end - sdy_start);
        return;
    }
        
    // 原图坐标
    const int dx = start_x + x;
    const int dy = start_y + y;

    if(dx >= width || dy >= height) 
        return;

    // 读取像素
    const int src_index = dy * width + dx;
    const uchar3 pixel = image[src_index];

    // 写入切片
    const int dst_index = slice_idx * slice_width * slice_height + y * slice_width + x;
    outs[dst_index] = pixel;
}

static void slice_plane(const uint8_t* image,
    uint8_t* outs,
    int* slice_start_point,
    const int width,
    const int height,
    const int slice_width,
    const int slice_height,
    const int slice_num_h,
    const int slice_num_v,
    void* stream=nullptr)
{
    int slice_total = slice_num_h * slice_num_v;
    hipStream_t stream_ = (hipStream_t)stream;
    dim3 block(32, 32);
    dim3 grid(
        (slice_width + block.x - 1) / block.x,
        (slice_height + block.y - 1) / block.y,
        slice_total
    );
    slice_kernel<<<grid, block, 0, stream_>>>(
        reinterpret_cast<const uchar3*>(image),
        reinterpret_cast<uchar3*>(outs),
        width, height, 
        slice_width, slice_height, 
        slice_num_h, slice_num_v, 
        slice_start_point
    );
}


namespace slice
{

int calculateNumCuts(int dimension, int subDimension, float overlapRatio) {
    float step = subDimension * (1 - overlapRatio);
    if (step == 0)
    {
        return 1;
    }
    float cuts = static_cast<float>(dimension - subDimension) / step;
    // 浮点数会有很小的误差，直接向上取整会出现多裁剪了一张图的情况
    if (fabs(cuts - round(cuts)) < 0.0001) {
        cuts = round(cuts);
    }
    int numCuts = static_cast<int>(std::ceil(cuts));
    return numCuts + 1;
}

static int calc_resolution_factor(int resolution)
{
    int expo = 0;
    while(pow(2, expo) < resolution) expo++;
    return expo - 1;
} 

static std::string calc_aspect_ratio_orientation(int width, int height)
{
    if (width < height)
        return  "vertical";
    else if(width > height)
        return "horizontal";
    else
        return "square";
}

static std::tuple<int, int, float, float> calc_ratio_and_slice(const std::string& orientation, int slide=1, float ratio=0.1)
{
    int slice_row, slice_col;
    float overlap_height_ratio, overlap_width_ratio;
    if (orientation == "vertical")
    {
        slice_row = slide;
        slice_col = slide * 2;
        overlap_height_ratio = ratio;
        overlap_width_ratio = ratio;
    }
    else if (orientation == "horizontal")
    {
        slice_row = slide * 2;
        slice_col = slide;
        overlap_height_ratio = ratio;
        overlap_width_ratio = ratio;
    }
    else if (orientation == "square")
    {
        slice_row = slide;
        slice_col = slide;
        overlap_height_ratio = ratio;
        overlap_width_ratio = ratio;
    }
    return std::make_tuple(slice_row, slice_col, overlap_height_ratio, overlap_width_ratio);
}

static std::tuple<int, int, float, float> calc_slice_and_overlap_params(
    const std::string& resolution, int width, int height, std::string orientation)
{
    int split_row, split_col;
    float overlap_height_ratio, overlap_width_ratio;
    if (resolution == "medium")
        std::tie(split_row, split_col, overlap_height_ratio, overlap_width_ratio) = calc_ratio_and_slice(
            orientation, 1, 0.8
        );

    else if (resolution == "high")
        std::tie(split_row, split_col, overlap_height_ratio, overlap_width_ratio) = calc_ratio_and_slice(
            orientation, 2, 0.4
        );

    else if (resolution == "ultra-high")
        std::tie(split_row, split_col, overlap_height_ratio, overlap_width_ratio) = calc_ratio_and_slice(
            orientation, 4, 0.4
        );
    else
    {
        split_col = 1;
        split_row = 1;
        overlap_width_ratio = 1;
        overlap_height_ratio = 1;
    }
    int slice_height = height / split_col;
    int slice_width = width / split_row;
    return std::make_tuple(slice_width, slice_height, overlap_height_ratio, overlap_width_ratio);
}

static std::tuple<int, int, float, float> get_resolution_selector(const std::string& resolution, int width, int height)
{
    std::string orientation = calc_aspect_ratio_orientation(width, height);
    return calc_slice_and_overlap_params(resolution, width, height, orientation);

}

static std::tuple<int, int, float, float> get_auto_slice_params(int width, int height)
{
    int resolution = height * width;
    int factor = calc_resolution_factor(resolution);
    if (factor <= 18)
        return get_resolution_selector("low", width, height);
    else if (18 <= factor && factor < 21)
        return get_resolution_selector("medium", width, height);
    else if (21 <= factor && factor < 24)
        return get_resolution_selector("high", width, height);
    else
        return get_resolution_selector("ultra-high", width, height);
}

std::vector<SlicedImageData> SliceImage::autoSlice(
        const tensor::Image& image,
        void* stream)
{
    int slice_width;
    int slice_height;
    float overlap_width_ratio;
    float overlap_height_ratio;
    std::tie(slice_width, slice_height, overlap_width_ratio, overlap_height_ratio) = get_auto_slice_params(image.width, image.height);
    return slice(image, slice_width, slice_height, overlap_width_ratio, overlap_height_ratio, stream);
}

std::vector<SlicedImageData> SliceImage::slice(
        const tensor::Image& image, 
        const int slice_width,
        const int slice_height,
        const float overlap_width_ratio,
        const float overlap_height_ratio,
        void* stream)
{
    slice_width_  = slice_width;
    slice_height_ = slice_height;
    hipStream_t stream_ = (hipStream_t)stream;

    int width = image.width;
    int height = image.height;

    slice_num_h_ = calculateNumCuts(width, slice_width, overlap_width_ratio);
    slice_num_v_ = calculateNumCuts(height, slice_height, overlap_height_ratio);
    printf("------------------------------------------------------\n"
            "CUDA SAHI CROP IMAGE ✂️\n"
            "Slice width                : %d\n"
            "Slice Height               : %d\n"
            "Overlap width  ratio       : %f\n"
            "Overlap height ratio       : %f\n"
            "Number of horizontal cuts  : %d\n"
            "Number of vertical cuts    : %d\n"
            "------------------------------------------------------\n", 
            slice_width, slice_height, overlap_width_ratio, overlap_height_ratio, slice_num_h_, slice_num_v_);
    int slice_num            = slice_num_h_ * slice_num_v_;
    int overlap_width_pixel  = slice_width  * overlap_width_ratio;
    int overlap_height_pixel = slice_height * overlap_height_ratio;
    // int slice_width   = (width - overlap_pixel) / slice_num_h + overlap_pixel;
    // int slice_height  = (height - overlap_pixel) / slice_num_v + overlap_pixel;

    size_t size_image = 3 * width * height;
    size_t output_img_size = 3 * slice_width * slice_height;

    input_image_.gpu(size_image);
    // input_image_.cpu(size_image);

    output_images_.gpu(slice_num * output_img_size);
    // output_images_.cpu(slice_num * output_img_size);

    checkRuntime(hipMemsetAsync(output_images_.gpu(), 114, output_images_.gpu_bytes(), stream_));
    slice_position_.resize(slice_num * 2);

    checkRuntime(hipMemcpyAsync(input_image_.gpu(), image.bgrptr, size_image, hipMemcpyHostToDevice, stream_));
    checkRuntime(hipStreamSynchronize(stream_));

    uint8_t* input_device = input_image_.gpu();
    uint8_t* output_device = output_images_.gpu();

    slice_start_point_.cpu(slice_num * 2);
    int* slice_start_point_ptr = slice_start_point_.cpu();
    
    for (int i = 0; i < slice_num_h_; i++)
    {
        int x = std::max(0, i * (slice_width - overlap_width_pixel));
        for (int j = 0; j < slice_num_v_; j++)
        {
            int y = std::max(0, j * (slice_height - overlap_height_pixel));
            int index = i * slice_num_v_ + j;
            slice_start_point_ptr[index*2]   = x;
            slice_start_point_ptr[index*2+1] = y;
            printf("i, j:%d,%d \t x,y : %d, %d\n", i, j, slice_start_point_ptr[index*2], slice_start_point_ptr[index*2+1]);
        } 
    }
    slice_start_point_.gpu(slice_num * 2);
    checkRuntime(hipMemcpyAsync(slice_start_point_.gpu(), slice_start_point_.cpu(), slice_num * 2 * sizeof(int), hipMemcpyHostToDevice, stream_));
    checkRuntime(hipStreamSynchronize(stream_));
    slice_plane(
        input_device, output_device, slice_start_point_.gpu(),
        width, height, 
        slice_width, slice_height, 
        slice_num_h_, slice_num_v_,
        stream);

    checkRuntime(hipStreamSynchronize(stream_));
    
    std::vector<SlicedImageData> slicedData(slice_num);
    for (int i = 0; i < slice_num; ++i) {
        slicedData[i].image = cv::Mat::zeros(slice_height, slice_width, CV_8UC3);
        slicedData[i].x = 0.0f;
        slicedData[i].y = 0.0f;
    }

    
    for (int i = 0; i < slice_num_h_; i++)
    {
        for (int j = 0; j < slice_num_v_; j++)
        {
            int index = i * slice_num_v_ + j;
            slice_position_[index*2]   = slice_start_point_ptr[index*2];
            slice_position_[index*2+1] = slice_start_point_ptr[index*2+1];
            slicedData[index].x = slice_start_point_ptr[index*2];
            slicedData[index].y = slice_start_point_ptr[index*2+1];
            slicedData[index].w = slice_width;
            slicedData[index].h = slice_height;
            uint8_t* output_img_data = slicedData[index].image.ptr<uint8_t>();
            hipMemcpyAsync(output_img_data, output_device+index*output_img_size, output_img_size*sizeof(uint8_t), hipMemcpyDeviceToHost, stream_);
        }
    }
    checkRuntime(hipStreamSynchronize(stream_));
    return slicedData;
}

}